#include "hip/hip_runtime.h"
#include "../../Engine/xdcore/precompiled.h"
#pragma hdrstop
#include "CUDA.h"

__global__ void knAdd(float *dst, const float *src0, const float *src1, const int count) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < count) {
		dst[idx] = src0[idx] + src1[idx];
	}
}

void CUDA_Add(float *dst, const float *src0, const float *src1, const int count) {
	dim3 grid((unsigned int)ceilf(count / 256.0f));
	dim3 block(256);
	knAdd<<<grid, block>>>(dst, src0, src1, count);	
}

__global__ void knSub(float *dst, const float *src0, const float *src1, const int count) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < count) {
		dst[idx] = src0[idx] - src1[idx];
	}
}

void CUDA_Sub(float *dst, const float *src0, const float *src1, const int count) {
	dim3 grid((unsigned int)ceilf(count / 256.0f));
	dim3 block(256);
	knSub<<<grid, block>>>(dst, src0, src1, count);	
}

__global__ void knMul(float *dst, const float *src0, const float *src1, const int count) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < count) {
		dst[idx] = src0[idx] * src1[idx];
	}
}

void CUDA_Mul(float *dst, const float *src0, const float *src1, const int count) {
	dim3 grid((unsigned int)ceilf(count / 256.0f));
	dim3 block(256);
	knMul<<<grid, block>>>(dst, src0, src1, count);	
}

__global__ void knDiv(float *dst, const float *src0, const float *src1, const int count) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < count) {
		dst[idx] = src0[idx] / src1[idx];
	}
}

void CUDA_Div(float *dst, const float *src0, const float *src1, const int count) {
	dim3 grid((unsigned int)ceilf(count / 256.0f));
	dim3 block(256);
	knDiv<<<grid, block>>>(dst, src0, src1, count);	
}

// FIXME!!
__global__ void knSum(float *dst, const float *src, const int count) {
	int gidx = blockDim.x * blockIdx.x + threadIdx.x;
	int tidx = threadIdx.x;

	__shared__ float partialSum[256];
	partialSum[tidx] = src[gidx];

	for (unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
		__syncthreads();
		if (tidx < stride) {
			partialSum[tidx] += partialSum[tidx + stride];
		}
	}

	if (tidx == 0) {
		dst[blockIdx.x] = partialSum[0];
	}
}

void CUDA_Sum(float *dst, const float *src, const int count) {
	dim3 grid(1);
	dim3 block(256);
	knSum<<<grid, block>>>(dst, src, count);	
}

template<int BLOCK_SIZE>
__global__ void knMatrixMultiply(float *dst, const float *src0, const float *src1, const int width) {
#if 1
	// shared memory �� ���� BLOCK_SIZE x BLOCK_SIZE ũ���� sub matrix
	__shared__ float as[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float bs[BLOCK_SIZE][BLOCK_SIZE];

	// matrix �� ��, �� index
	const int row_idx = BLOCK_SIZE * blockIdx.y + threadIdx.y;
	const int col_idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;	

	// thread �� sub matrix �� ���ҵ� offset
	const int oa = width * row_idx + threadIdx.x;
	const int ob = width * threadIdx.y + col_idx;

	// ù��° sub matrix �� ���ҵ��� prefetch
	float a = src0[oa];
	float b = src1[ob];

	// block stride �� �ִ� ũ��
	const int end = BLOCK_SIZE * gridDim.x;

	// ��� ��
	float comp = 0.0f;

	for (int stride = BLOCK_SIZE; stride <= end; stride += BLOCK_SIZE) {
		// thread ���� prefetch �� ���ҵ��� shared memory �� ����
		as[threadIdx.y][threadIdx.x] = a;
		bs[threadIdx.y][threadIdx.x] = b;

		// ���� block �� shared memory ���� ���
		__syncthreads();

		// ���� sub matrix �� ���ҵ��� prefetch
		a = src0[oa + stride];
		b = src1[ob + width * stride];

		// sub matrix �� �̿��� dot product
		for (int i = 0; i < BLOCK_SIZE; i += 4) {
			comp += as[threadIdx.y][i + 0] * bs[i + 0][threadIdx.x];
			comp += as[threadIdx.y][i + 1] * bs[i + 1][threadIdx.x];
			comp += as[threadIdx.y][i + 2] * bs[i + 2][threadIdx.x];
			comp += as[threadIdx.y][i + 3] * bs[i + 3][threadIdx.x];
		}

		// ���� sub matrix �� block �� ���ؼ� ���
		__syncthreads();
	}

	// global memory �� ��� �� ����
	dst[width * row_idx + col_idx] = comp;
#else
	const int col_idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	const int row_idx = BLOCK_SIZE * blockIdx.y + threadIdx.y;	

	float comp = 0.0f;
	for (int i = 0; i < width; i++) {
		comp += src0[width * row_idx + i] * src1[width * i + col_idx];
	}
	dst[width * row_idx + col_idx] = comp;
#endif
}

template<int BLOCK_SIZE>
__global__ void knMatrixMultiply2(float *dst, const float *src0, const float *src1, const int width) {
	__shared__ float as[BLOCK_SIZE][2*BLOCK_SIZE];
	__shared__ float bs[2*BLOCK_SIZE][BLOCK_SIZE];

	int col_idx = 2*BLOCK_SIZE * blockIdx.x + threadIdx.x;
	int row_idx = BLOCK_SIZE * blockIdx.y + threadIdx.y;

	int oa = width * row_idx + threadIdx.x;
	int ob = width * threadIdx.y + col_idx;

	float a00 = src0[oa];
	float a01 = src0[oa + BLOCK_SIZE];

	float b00 = src1[ob];
	float b01 = src1[ob + BLOCK_SIZE];
	float b10 = src1[ob + width * BLOCK_SIZE];
	float b11 = src1[ob + width * BLOCK_SIZE + BLOCK_SIZE];

	int end = 2*BLOCK_SIZE * gridDim.x;

	float bcomp00 = 0.0f;
	float bcomp01 = 0.0f;
	
	for (int stride = 2*BLOCK_SIZE; stride <= end; stride += 2*BLOCK_SIZE) {
		as[threadIdx.y][threadIdx.x] = a00;
		as[threadIdx.y][threadIdx.x + BLOCK_SIZE] = a01;
		
		bs[threadIdx.y][threadIdx.x] = b00;
		bs[threadIdx.y + BLOCK_SIZE][threadIdx.x] = b10;
		
		__syncthreads();

		a00 = src0[oa + stride];
		a01 = src0[oa + BLOCK_SIZE + stride];
		
		b00 = src1[ob + width * stride];
		b10 = src1[ob + width * (BLOCK_SIZE + stride)];
		
		for (int i = 0; i < 2*BLOCK_SIZE; i += 4) {
			bcomp00 += as[threadIdx.y][i + 0] * bs[i + 0][threadIdx.x];
			bcomp00 += as[threadIdx.y][i + 1] * bs[i + 1][threadIdx.x];
			bcomp00 += as[threadIdx.y][i + 2] * bs[i + 2][threadIdx.x];
			bcomp00 += as[threadIdx.y][i + 3] * bs[i + 3][threadIdx.x];
		}

		__syncthreads();

		bs[threadIdx.y][threadIdx.x] = b01;
		bs[threadIdx.y + BLOCK_SIZE][threadIdx.x] = b11;

		__syncthreads();
				
		b01 = src1[ob + width * stride + BLOCK_SIZE];
		b11 = src1[ob + width * (BLOCK_SIZE + stride) + BLOCK_SIZE];

		for (int i = 0; i < 2*BLOCK_SIZE; i += 4) {
			bcomp01 += as[threadIdx.y][i + 0] * bs[i + 0][threadIdx.x];
			bcomp01 += as[threadIdx.y][i + 1] * bs[i + 1][threadIdx.x];
			bcomp01 += as[threadIdx.y][i + 2] * bs[i + 2][threadIdx.x];
			bcomp01 += as[threadIdx.y][i + 3] * bs[i + 3][threadIdx.x];
		}
		
		__syncthreads();
	}

	dst[width * row_idx + col_idx] = bcomp00;
	dst[width * row_idx + col_idx + BLOCK_SIZE] = bcomp01;	
}

void CUDA_MatrixMultiply(float *dst, const float *src0, const float *src1, const int width) {
#if 1
	if (MyCuda::deviceProp[0].maxThreadsPerBlock >= 1024) {
		dim3 grid((width + 31) / 32, (width + 31) / 32);
		dim3 block(32, 32);
		knMatrixMultiply<32><<<grid, block>>>(dst, src0, src1, width);
	} else {
		dim3 grid((width + 15) / 16, (width + 15) / 16);
		dim3 block(16, 16);
		knMatrixMultiply<16><<<grid, block>>>(dst, src0, src1, width);
	}
#else
	dim3 grid((width + 31) / 32, (width + 15) / 16);
	dim3 block(16, 16);
	knMatrixMultiply2<16><<<grid, block>>>(dst, src0, src1, width);
#endif
}